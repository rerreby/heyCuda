#include <stdio.h>

/*
void getComputeModeDescription(int computeMode, **char result)
{
    if (computeMode == 0)
        &result = "Thrreading mode available";
    else
        &result = "Unknown";
}
*/

int main(int argc, char **argv)
{
    printf("Hey, Cuda! utility, v0.1\n\n");

    int device;
    hipDeviceProp_t prop;


    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    printf("Device Name: %s\n", prop.name);
    printf("Processors: %d\n", prop.multiProcessorCount);
    printf("Clock Rate: %d (Mhz)\n", prop.clockRate / 1024);
    printf("Total Memory: %d Megabytes\n", (prop.totalGlobalMem / 1024) / 1024);
    printf("Memory Clock Rate: %d Mhz\n", (prop.memoryClockRate / 1024));
    printf("Memory BUS width: %d bits\n", prop.memoryBusWidth);
    printf("CUDA compute mode: %d\n", prop.computeMode);
    printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Registers per Multiprocessor: %d\n", prop.regsPerMultiprocessor);
    printf("PCI Device ID: %d\n", prop.pciDeviceID);

    printf("\n");

    printf("Supports allocating managed memory: %s\n", (prop.managedMemory == 0) ? "No" : "Yes");
    printf("Device is %s\n", (prop.integrated == 0) ? "discrete" : "integrated");

    // release device
    hipDeviceSynchronize();
    return 0;
}